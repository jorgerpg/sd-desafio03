// saxpy_cuda.cu
// Compile: nvcc -O3 -std=c++17 saxpy_cuda.cu -o saxpy_cuda
// Run:     ./saxpy_cuda [N] [use_pinned]
// Ex.:     ./saxpy_cuda 50000000 1    // 50M elementos, host pinned (mais rápido nas cópias)

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) do { \
  hipError_t err = (call); \
  if (err != hipSuccess) { \
    std::fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
    std::exit(EXIT_FAILURE); \
  } \
} while(0)

__global__ void saxpy_kernel(const float a, const float* __restrict__ x, float* __restrict__ y, size_t n) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a * x[i] + y[i];
  }
}

int main(int argc, char** argv) {
  size_t N = (argc > 1) ? std::stoull(argv[1]) : 50'000'000ULL;
  bool use_pinned = (argc > 2) ? (std::atoi(argv[2]) != 0) : false;
  const float a = 2.0f;

  std::cout << "CUDA | N=" << N << " | host=" << (use_pinned ? "pinned" : "pageable") << "\n";

  float *h_x = nullptr, *h_y = nullptr, *h_y_out = nullptr;
  if (use_pinned) {
    CUDA_CHECK(hipHostMalloc(&h_x,     N*sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc(&h_y,     N*sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc(&h_y_out, N*sizeof(float), hipHostMallocDefault));
  } else {
    h_x     = (float*)std::malloc(N*sizeof(float));
    h_y     = (float*)std::malloc(N*sizeof(float));
    h_y_out = (float*)std::malloc(N*sizeof(float));
  }
  if (!h_x || !h_y || !h_y_out) { std::cerr << "Falha ao alocar host.\n"; return 1; }

  // Inicialização determinística
  for (size_t i = 0; i < N; ++i) {
    h_x[i] = (float)((i % 1000) - 500) / 500.0f;
    h_y[i] = (float)((i % 777) - 388) / 388.0f;
  }

  float *d_x=nullptr, *d_y=nullptr;
  CUDA_CHECK(hipMalloc(&d_x, N*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_y, N*sizeof(float)));

  hipEvent_t ev_start, ev_after_h2d, ev_after_kernel, ev_end;
  CUDA_CHECK(hipEventCreate(&ev_start));
  CUDA_CHECK(hipEventCreate(&ev_after_h2d));
  CUDA_CHECK(hipEventCreate(&ev_after_kernel));
  CUDA_CHECK(hipEventCreate(&ev_end));

  // Warm-up
  CUDA_CHECK(hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_y, h_y, N*sizeof(float), hipMemcpyHostToDevice));
  
  const int block = 256;
  const int grid  = (int)std::min((size_t)131072, (N + block - 1) / block);
  saxpy_kernel<<<grid, block>>>(a, d_x, d_y, N);
  CUDA_CHECK(hipDeviceSynchronize());

  // Benchmark
  CUDA_CHECK(hipEventRecord(ev_start));
  CUDA_CHECK(hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_y, h_y, N*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipEventRecord(ev_after_h2d));

  saxpy_kernel<<<grid, block>>>(a, d_x, d_y, N);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipEventRecord(ev_after_kernel));

  CUDA_CHECK(hipMemcpy(h_y_out, d_y, N*sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipEventRecord(ev_end));
  CUDA_CHECK(hipEventSynchronize(ev_end));

  float ms_total=0, ms_h2d=0, ms_kernel=0, ms_d2h=0;
  CUDA_CHECK(hipEventElapsedTime(&ms_total, ev_start, ev_end));
  CUDA_CHECK(hipEventElapsedTime(&ms_h2d,   ev_start, ev_after_h2d));
  CUDA_CHECK(hipEventElapsedTime(&ms_kernel,ev_after_h2d, ev_after_kernel));
  CUDA_CHECK(hipEventElapsedTime(&ms_d2h,   ev_after_kernel, ev_end));

  // Verificação rápida de erro contra referência CPU simples (amostra)
  double max_abs_err = 0.0;
  for (int k = 0; k < 1000 && (size_t)k < N; ++k) {
    float ref = a * h_x[k] + h_y[k];
    max_abs_err = std::max(max_abs_err, (double)std::abs(ref - h_y_out[k]));
  }

  // Métricas
  const double flops = 2.0 * (double)N;     // 1 mult + 1 soma
  const double bytes = 12.0 * (double)N;    // ler x (4), ler y (4), escrever y (4)

  double gflops_kernel = (flops / 1e9) / (ms_kernel / 1e3);
  double gbps_total    = (bytes / 1e9) / (ms_total / 1e3);

  std::cout << "H2D: " << ms_h2d << " ms | K: " << ms_kernel << " ms | D2H: " << ms_d2h 
            << " ms | Total: " << ms_total << " ms\n";
  std::cout << "Kernel: " << gflops_kernel << " GFLOP/s | Total BW ~ " << gbps_total << " GB/s\n";
  std::cout << "Max |err| (amostra 1k): " << max_abs_err << "\n";

  CUDA_CHECK(hipFree(d_x));
  CUDA_CHECK(hipFree(d_y));
  CUDA_CHECK(hipEventDestroy(ev_start));
  CUDA_CHECK(hipEventDestroy(ev_after_h2d));
  CUDA_CHECK(hipEventDestroy(ev_after_kernel));
  CUDA_CHECK(hipEventDestroy(ev_end));

  if (use_pinned) {
    CUDA_CHECK(hipHostFree(h_x));
    CUDA_CHECK(hipHostFree(h_y));
    CUDA_CHECK(hipHostFree(h_y_out));
  } else {
    std::free(h_x);
    std::free(h_y);
    std::free(h_y_out);
  }
  return 0;
}